#include "hip/hip_runtime.h"
// Standard C++ headers
#include <iostream>
#include <vector>
#include <stdexcept>
#include <cmath>
#include <cassert>
#include <chrono>
#include <algorithm>


// CUDA headers
#include <hip/hip_runtime.h>
#include "../include/cudaerr.h"
#include "../include/kernels.cu"

// Custom headers
#include "../include/loadbin.h"
#include "../include/savebin.h"
#include "../include/inverse.h"


// Define the wavelet coefficients as floats
// Low coefficients
const std::vector<std::vector<float>> db_low = {
    {0.70710678f, 0.70710678f}, // db1
    {-0.12940952f, 0.22414387f, 0.83651630f, 0.48296291f}, // db2
    {0.03522629f, -0.08544127f, -0.13501102f, 0.45987750f, 0.80689151f, 0.33267055f}, // db3
    {-0.01059740f, 0.03288301f, 0.03084138f, -0.18703481f, -0.02798377f, 0.63088077f, 0.71484657f, 0.23037781f} // db4
};
// High coefficients
const std::vector<std::vector<float>> db_high = {
    {-0.70710678f, 0.70710678f}, // db1
    {-0.48296291f, 0.83651630f, -0.22414387f, -0.12940952f}, // db2
    {-0.33267055f, 0.80689151f, -0.45987750f, -0.13501102f, 0.08544127f, 0.03522629f}, // db3
    {-0.23037781f, 0.71484657f, -0.63088077f, -0.02798377f, 0.18703481f, 0.03084138f, -0.03288301f, -0.01059740f} // db4
};

void toGPU(std::vector<std::vector<std::vector<float>>> volume, int db_num, int depth, int rows, int cols, float*& d_low_coeff, float*& d_high_coeff, int&filter_size, float*& d_volume) {

    // Select the coefficients based on db_num
    std::vector<float> low_coeff = db_low[db_num - 1];
    std::vector<float> high_coeff = db_high[db_num - 1];

    // Calculate the filter size
    filter_size = low_coeff.size();

    // Allocate memory for the low and high pass coefficients on the GPU
    hipError_t err = hipMalloc(&d_low_coeff, low_coeff.size() * sizeof(float));
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate GPU memory for low coefficients: " + std::string(hipGetErrorString(err)));
    }

    err = hipMalloc(&d_high_coeff, high_coeff.size() * sizeof(float));
    if (err != hipSuccess) {
        hipFree(d_low_coeff); // Free previously allocated memory
        throw std::runtime_error("Failed to allocate GPU memory for high coefficients: " + std::string(hipGetErrorString(err)));
    }

    // Copy the coefficients to the GPU
    err = hipMemcpy(d_low_coeff, low_coeff.data(), low_coeff.size() * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(d_low_coeff);
        hipFree(d_high_coeff);
        throw std::runtime_error("Failed to copy low coefficients to GPU: " + std::string(hipGetErrorString(err)));
    }

    err = hipMemcpy(d_high_coeff, high_coeff.data(), high_coeff.size() * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(d_low_coeff);
        hipFree(d_high_coeff);
        throw std::runtime_error("Failed to copy high coefficients to GPU: " + std::string(hipGetErrorString(err)));
    }

    // Flatten the 3D volume into a 1D vector (row-major order)
    std::vector<float> flat_volume(depth * rows * cols);
    for (int d = 0; d < depth; ++d) {
        for (int r = 0; r < rows; ++r) {
            for (int c = 0; c < cols; ++c) {
                flat_volume[d * rows * cols + r * cols + c] = volume[d][r][c];
            }
        }
    }

    // Allocate memory on the GPU for the volume
    err = hipMalloc(&d_volume, flat_volume.size() * sizeof(float));
    if (err != hipSuccess) {
        hipFree(d_low_coeff);
        hipFree(d_high_coeff);
        throw std::runtime_error("Failed to allocate GPU memory for volume: " + std::string(hipGetErrorString(err)));
    }

    // Copy the flattened volume to the GPU
    err = hipMemcpy(d_volume, flat_volume.data(), flat_volume.size() * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(d_low_coeff);
        hipFree(d_high_coeff);
        hipFree(d_volume);
        throw std::runtime_error("Failed to copy volume data to GPU: " + std::string(hipGetErrorString(err)));
    }

    // Clear the CPU memory after copying to GPU
    flat_volume.clear();
    volume.clear();

    // Synchronize the device with error checking
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to synchronize the device: " + std::string(hipGetErrorString(err)));
    }
}

std::vector<std::vector<std::vector<float>>> volCPU(float* d_volume, int depth, int rows, int cols) {
    // Allocate memory for the 3D volume on the CPU
    std::vector<std::vector<std::vector<float>>> volume(depth, std::vector<std::vector<float>>(rows, std::vector<float>(cols)));

    // Copy the data from the GPU to the CPU
    std::vector<float> flat_volume(depth * rows * cols);
    hipError_t err = hipMemcpy(flat_volume.data(), d_volume, flat_volume.size() * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to copy data from GPU to CPU: " + std::string(hipGetErrorString(err)));
    }

    // Unflatten the 1D vector into a 3D volume
    for (int d = 0; d < depth; ++d) {
        for (int r = 0; r < rows; ++r) {
            for (int c = 0; c < cols; ++c) {
                volume[d][r][c] = flat_volume[(d * rows * cols) + (r * cols) + c];
            }
        }
    }

    // Free the allocated GPU memory
    err = hipFree(d_volume);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to free GPU memory: " + std::string(hipGetErrorString(err)));
    }

    return volume;
}

__global__ void reduce(float* volume, float* temp, int depth, int rows, int cols) {
    int d = blockIdx.z * blockDim.z + threadIdx.z;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (d < depth / 2 && r < rows / 2 && c < cols / 2) {
        // Calculate indices for LLL subband
        int old_idx = d * (rows / 2) * (cols / 2) + r * (cols / 2) + c;
        int new_idx = d * rows * cols + r * cols + c;

        // Extract LLL subband
        temp[new_idx] = volume[old_idx];
    }
}

__global__ void copy_transformed_data(float* d_transformed, float* d_final, size_t depth, size_t rows, size_t cols, size_t orig_depth, size_t orig_rows, size_t orig_cols) {
    int d = blockIdx.z * blockDim.z + threadIdx.z;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (d < depth && r < rows && c < cols) {
        d_final[d * orig_rows * orig_cols + r * orig_cols + c] = d_transformed[d * rows * cols + r * cols + c];
    }
}


void dwt_3d(float* d_volume, float* d_low_coeff, float* d_high_coeff, int depth, int rows, int cols, int filter_size) {
    float* d_data1 = d_volume;
    float* d_data2 = nullptr;
    hipError_t err = hipMalloc(&d_data2, depth * rows * cols * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate GPU memory for temporary volume: " << hipGetErrorString(err) << std::endl;
    }

    dim3 blockDim(16, 8, 8);
    dim3 gridDim0((rows + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y, (depth + blockDim.z - 1) / blockDim.z);
    dim3 gridDim1((cols + blockDim.x - 1) / blockDim.x, (rows + blockDim.y - 1) / blockDim.y, (depth + blockDim.z - 1) / blockDim.z);
    dim3 gridDim2((depth + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y, (rows + blockDim.z - 1) / blockDim.z);


    // Perform convolution along the first dimension
    dim0_kernel<<<gridDim0, blockDim, filter_size * sizeof(float) * 2>>>(d_data1, d_data2, d_low_coeff, d_high_coeff, filter_size, depth, rows, cols);
    // Synchronize the device
    hipDeviceSynchronize();


    // Perform convolution along the second dimension
    dim1_kernel<<<gridDim1, blockDim, filter_size * sizeof(float) * 2>>>(d_data2, d_data1, d_low_coeff, d_high_coeff, filter_size, depth, rows, cols);
    // Synchronize the device  
    hipDeviceSynchronize();

    // Perform convolution along the third dimension
    dim2_kernel<<<gridDim2, blockDim, filter_size * sizeof(float) * 2>>>(d_data1, d_data2, d_low_coeff, d_high_coeff, filter_size, depth, rows, cols);
    // Synchronize the device
    hipDeviceSynchronize();



    // Copy the transformed data back to the original volume
    dim3 gridDim((cols + blockDim.x - 1) / blockDim.x, (rows + blockDim.y - 1) / blockDim.y, (depth + blockDim.z - 1) / blockDim.z);
    copy_transformed_data<<<gridDim, blockDim>>>(d_data2, d_volume, depth, rows, cols, depth, rows, cols);
    // Synchronize the device
    hipDeviceSynchronize();

    // Free the temporary volume
    err = hipFree(d_data2);
    if (err != hipSuccess) {
        std::cerr << "Failed to free GPU memory for temporary volume: " << hipGetErrorString(err) << std::endl;
    }
}


void multi_level(float* d_volume, float* d_low_coeff, float* d_high_coeff, int levels, int& depth, int& rows, int& cols, int filter_size) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    
    for (int i = 0; i < levels; i++) {
        dwt_3d(d_volume, d_low_coeff, d_high_coeff, depth, rows, cols, filter_size);
        
        if (i == levels - 1) {break;}
        
    }
    

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Multi-level DWT completed in " << milliseconds << " ms" << std::endl;

    hipEventDestroy(start);
    hipEventDestroy(stop);
}

// Main program entry point
int main(int argc, char *argv[]) {
    (void)argc; // Suppress unused parameter warning
    // Print the program title
    std::cerr << "Assignment 2: CUDA Implementation of 3D DWT" << std::endl;

    // Check if the number of arguments is correct
    assert(argc == 6 && "Usage: ./assignment-2 <input.bin> <output.bin> <db_num> <levels> <inverse>");

    // Start the global timer
    auto start = std::chrono::high_resolution_clock::now();

    // Load the arguments into variables
    std::string bin_in = argv[1];
    std::string bin_out = argv[2];
    int db_num = std::stoi(argv[3]);
    int levels = std::stoi(argv[4]);
    int inverse = std::stoi(argv[5]);

    // Load the 3D volume from the binary file
    std::vector<std::vector<std::vector<float>>> vol_in = loadvolume(bin_in);

    // Get the dimensions of the 3D volume
    int depth = vol_in.size();
    int rows = vol_in[0].size();
    int cols = vol_in[0][0].size();

    // Print the dimensions of the 3D volume
    std::cerr << "Volume dimensions: " << depth << "x" << rows << "x" << cols << std::endl;

    // Define volume for the output
    std::vector<std::vector<std::vector<float>>> vol_out;

    // Perform the inverse if inverse flag is set to 1
    if (inverse == 1) {
        std::cerr << "Performing inverse DWT" << std::endl;

        inverse_multi_level(vol_in, db_num, levels);

        // Swap vol_in and name it vol_out
        vol_out = vol_in;

        // print the dimensions of volume after inverse DWT
        std::cerr << "Volume dimensions after inverse DWT: " << vol_out.size() << "x" << vol_out[0].size() << "x" << vol_out[0][0].size() << std::endl;

    }
    else{
        
        // Create pointers for the volume and coefficients on the GPU
        float* d_low_coeff = nullptr;
        float* d_high_coeff = nullptr;
        float* d_volume = nullptr;
        int filter_size;

        // Copy the volume and coefficients to the GPU
        toGPU(vol_in, db_num, depth, rows, cols, d_low_coeff, d_high_coeff, filter_size, d_volume);
        
        // Perform the multi-level DWT on the 3D volume
        multi_level(d_volume, d_low_coeff, d_high_coeff, levels, depth, rows, cols, filter_size);

        // Copy the data back to the CPU
        vol_out = volCPU(d_volume, depth, rows, cols);

        // print the dimensions of volume after DWT
        std::cerr << "Volume dimensions after DWT: " << vol_out.size() << "x" << vol_out[0].size() << "x" << vol_out[0][0].size() << std::endl;
    }

    // Save the modified 3D volume to the output binary file
    savevolume(vol_out, bin_out);

    // Stop the global timer
    auto end = std::chrono::high_resolution_clock::now();
    // Calculate the duration
    std::chrono::duration<double> t = end - start;

    // Log the time taken for the program
    std::cerr << "Total time taken: " << t.count() << " seconds" << std::endl;

    /*
    Fix the inverse transform so that the first slices match

    Try fix the multi level with regards to the cuda kernel

    Try make the segemented the memory allocations and the kernel calls to be more efficient
    
    */

    return EXIT_SUCCESS;
}
