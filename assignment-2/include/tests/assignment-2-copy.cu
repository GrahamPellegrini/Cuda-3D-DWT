#include "hip/hip_runtime.h"
// Standard C++ headers
#include <iostream>
#include <vector>
#include <stdexcept>
#include <cmath>
#include <cassert>
#include <chrono>

// CUDA headers
#include <hip/hip_runtime.h>

// Custom headers
#include "../include/loadbin.h"
#include "../include/savebin.h"


// Define the wavelet coefficients as floats
// Low coefficients
const std::vector<std::vector<float>> db_low = {
    {0.70710678f, 0.70710678f}, // db1
    {-0.12940952f, 0.22414387f, 0.83651630f, 0.48296291f}, // db2
    {0.03522629f, -0.08544127f, -0.13501102f, 0.45987750f, 0.80689151f, 0.33267055f}, // db3
    {-0.01059740f, 0.03288301f, 0.03084138f, -0.18703481f, -0.02798377f, 0.63088077f, 0.71484657f, 0.23037781f} // db4
};
// High coefficients
const std::vector<std::vector<float>> db_high = {
    {-0.70710678f, 0.70710678f}, // db1
    {-0.48296291f, 0.83651630f, -0.22414387f, -0.12940952f}, // db2
    {-0.33267055f, 0.80689151f, -0.45987750f, -0.13501102f, 0.08544127f, 0.03522629f}, // db3
    {-0.23037781f, 0.71484657f, -0.63088077f, -0.02798377f, 0.18703481f, 0.03084138f, -0.03288301f, -0.01059740f} // db4
};

__constant__ float d_low_coeff[8];   // Maximum size for low-pass filters
__constant__ float d_high_coeff[8]; // Maximum size for high-pass filters
__constant__ int d_filter_length;

void coeffGPU(int db_num) {
    // Ensure db_num is between 1 and 4
    assert(db_num >= 1 && db_num <= 4 && "db_num must be between 1 and 4");

    // Select the coefficients based on db_num
    std::vector<float> low_coeff = db_low[db_num - 1];
    std::vector<float> high_coeff = db_high[db_num - 1];

    // Fill the rest of the array with 0.0f if the filter is shorter than 8 coefficients
    std::vector<float> low_coeff_filled(8, 0.0f);
    std::vector<float> high_coeff_filled(8, 0.0f);

    // Copy the selected coefficients into the first part of the arrays
    std::copy(low_coeff.begin(), low_coeff.end(), low_coeff_filled.begin());
    std::copy(high_coeff.begin(), high_coeff.end(), high_coeff_filled.begin());

    // Copy the filled arrays to GPU's constant memory
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(d_low_coeff), low_coeff_filled.data(), 8 * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Error copying low coefficients to GPU: " << hipGetErrorString(err) << std::endl;
        return;
    }

    err = hipMemcpyToSymbol(HIP_SYMBOL(d_high_coeff), high_coeff_filled.data(), 8 * sizeof(float));
    if (err != hipSuccess) {
        std::cerr << "Error copying high coefficients to GPU: " << hipGetErrorString(err) << std::endl;
        return;
    }

    // Copy the filter length for the selected wavelet to GPU's constant memory
    int filter_length = low_coeff.size();
    err = hipMemcpyToSymbol(HIP_SYMBOL(d_filter_length), &filter_length, sizeof(int));
    if (err != hipSuccess) {
        std::cerr << "Error copying filter length to GPU: " << hipGetErrorString(err) << std::endl;
        return;
    }

    std::cerr << "Coefficients and filter length for db" << db_num << " successfully copied to GPU" << std::endl;
}

float* volGPU(const std::vector<std::vector<std::vector<float>>>& volume, int depth, int rows, int cols) {
    // Flatten the 3D volume into a 1D vector
    std::vector<float> flat_volume(depth * rows * cols);
    for (int d = 0; d < depth; ++d) {
        for (int r = 0; r < rows; ++r) {
            for (int c = 0; c < cols; ++c) {
                flat_volume[(d * rows * cols) + (r * cols) + c] = volume[d][r][c];
            }
        }
    }

    // Allocate memory on the GPU
    float* d_volume = nullptr;
    hipError_t err = hipMalloc(&d_volume, flat_volume.size() * sizeof(float));
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate GPU memory for volume: " + std::string(hipGetErrorString(err)));
    }

    // Copy data to the GPU
    err = hipMemcpy(d_volume, flat_volume.data(), flat_volume.size() * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        // Free the allocated memory in case of an error
        hipFree(d_volume);
        throw std::runtime_error("Failed to copy data to GPU: " + std::string(hipGetErrorString(err)));
    }

    return d_volume;
}

std::vector<std::vector<std::vector<float>>> volCPU(float* d_volume, int depth, int rows, int cols) {
    // Allocate memory for the 3D volume on the CPU
    std::vector<std::vector<std::vector<float>>> volume(depth, std::vector<std::vector<float>>(rows, std::vector<float>(cols)));

    // Copy the data from the GPU to the CPU
    std::vector<float> flat_volume(depth * rows * cols);
    hipError_t err = hipMemcpy(flat_volume.data(), d_volume, flat_volume.size() * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to copy data from GPU to CPU: " + std::string(hipGetErrorString(err)));
    }

    // Unflatten the 1D vector into a 3D volume
    for (int d = 0; d < depth; ++d) {
        for (int r = 0; r < rows; ++r) {
            for (int c = 0; c < cols; ++c) {
                volume[d][r][c] = flat_volume[(d * rows * cols) + (r * cols) + c];
            }
        }
    }

    // Free the allocated GPU memory
    err = hipFree(d_volume);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to free GPU memory: " + std::string(hipGetErrorString(err)));
    }

    return volume;
}

__global__ void dwt_1d_kernel(
    float* volume,
    int stride_outer,
    int stride_inner,
    int outer_size,
    int inner_size
) {
    // Calculate the outer index
    int outer_idx = blockIdx.x * blockDim.x + threadIdx.x;
    // If the outer index is out of bounds, return
    if (outer_idx >= outer_size){
        printf("Outer index out of bounds\n");
        return;
    }

    // Allocate shared memory for the temporary results
    extern __shared__ float temp[];  
    int filter_length = d_filter_length;
    int low_offset = 0;
    int high_offset = inner_size / 2;

    for (int i = 0; i <= inner_size - filter_length; i += 2) {
        float low_sum = 0.0f, high_sum = 0.0f;

        for (int j = 0; j < filter_length; ++j) {
            int idx = outer_idx * stride_outer + (i + j) * stride_inner;
            // If the index is out of bounds, skip the calculation
            if (idx < outer_size * stride_outer) {
                low_sum += volume[idx] * d_low_coeff[j];
                high_sum += volume[idx] * d_high_coeff[j];
            }
        }

        temp[low_offset + i / 2] = low_sum;
        temp[high_offset + i / 2] = high_sum;
    }

    __syncthreads();

    for (int i = 0; i < inner_size; ++i) {
        int idx = outer_idx * stride_outer + i * stride_inner;
        if (idx < outer_size * stride_outer) { // Prevent out-of-bounds access
            volume[idx] = temp[i];
        }
    }
}

void dwt_3d(float* volume, int depth, int rows, int cols) {
    // Define CUDA launch parameters
    dim3 block_dim(256); // Threads per block
    int shared_mem_size = cols * sizeof(float); // Shared memory size for each thread block

    // Step 1: Apply 1D DWT along rows for each depth slice
    dim3 grid_dim_rows((rows + block_dim.x - 1) / block_dim.x, depth);
    dwt_1d_kernel<<<grid_dim_rows, block_dim, shared_mem_size>>>(
        volume, cols, 1, depth * rows, cols);
    hipDeviceSynchronize();

    // Step 2: Apply 1D DWT along columns for each depth slice
    dim3 grid_dim_cols((cols + block_dim.x - 1) / block_dim.x, depth);
    dwt_1d_kernel<<<grid_dim_cols, block_dim, shared_mem_size>>>(
        volume, rows * cols, cols, depth * cols, rows);
    hipDeviceSynchronize();

    // Step 3: Apply 1D DWT along depth for each row-column combination
    dim3 grid_dim_depth((depth + block_dim.x - 1) / block_dim.x);
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            float* depth_ptr = volume + i * cols + j;
            dwt_1d_kernel<<<grid_dim_depth, block_dim, shared_mem_size>>>(
                depth_ptr, rows * cols, rows * cols, depth, 1);
            hipDeviceSynchronize();
        }
    }
}


void multi_level(float* d_volume, int levels, int& depth, int& rows, int& cols) {
    // Initialize CUDA timer events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    // Perform multi-level DWT
    for (int i = 0; i < levels; i++) {
        // Perform a single-level 3D DWT
        dwt_3d(d_volume, depth, rows, cols);

        // Update dimensions for the next level (reduce by half)
        depth = (depth + 1) / 2;
        rows = (rows + 1) / 2;
        cols = (cols + 1) / 2;

        // Ensure dimensions remain valid
        if (depth < 1 || rows < 1 || cols < 1) {
            throw std::runtime_error("Volume dimensions became invalid during multi-level decomposition.");
        }
    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Multi-level DWT completed in " << milliseconds << " ms" << std::endl;

    // Cleanup CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);
}



// Main program entry point
int main(int argc, char *argv[]) {
    (void)argc; // Suppress unused parameter warning
    // Print the program title
    std::cerr << "Assignment 2: CUDA Implementation of 3D DWT" << std::endl;

    // Check if the number of arguments is correct
    assert(argc == 5 && "Usage: ./assignment-1 <input.bin> <output.bin> <db_num> <levels>");

    // Start the global timer
    auto start = std::chrono::high_resolution_clock::now();

    // Load the arguments into variables
    std::string bin_in = argv[1];
    std::string bin_out = argv[2];
    int db_num = std::stoi(argv[3]);
    int levels = std::stoi(argv[4]);

    // Send the wavelet coefficients to the GPU
    coeffGPU(db_num);

    // Load the 3D volume from the binary file
    std::vector<std::vector<std::vector<float>>> volume = loadvolume(bin_in);

    // Get the dimensions of the 3D volume
    int depth = volume.size();
    int rows = volume[0].size();
    int cols = volume[0][0].size();

    // Allocate memory on the GPU for the 3D volume
    float* d_volume = volGPU(volume, depth, rows, cols);

    // Synchronize the device after memory transfer
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        std::cerr << "Error synchronizing the device: " << hipGetErrorString(err) << std::endl;
        return EXIT_FAILURE;
    }

    // Perform the multi-level DWT on the 3D volume
    multi_level(d_volume, levels, depth, rows, cols);

    // Copy the data back to the CPU
    volume = volCPU(d_volume, depth, rows, cols);

    // Save the modified 3D volume to the output binary file
    savevolume(volume, bin_out);

    // Stop the global timer
    auto end = std::chrono::high_resolution_clock::now();
    // Calculate the duration
    std::chrono::duration<double> t = end - start;

    // Log the time taken for the program
    std::cerr << "Total time taken: " << t.count() << " seconds" << std::endl;

    return EXIT_SUCCESS;
}