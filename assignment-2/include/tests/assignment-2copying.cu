#include "hip/hip_runtime.h"
// Standard C++ headers
#include <iostream>
#include <vector>
#include <stdexcept>
#include <cmath>
#include <cassert>
#include <chrono>
#include <algorithm>


// CUDA headers
#include <hip/hip_runtime.h>
#include "../include/cudaerr.h"
#include "../include/kernels.cu"

// Custom headers
#include "../include/loadbin.h"
#include "../include/savebin.h"


// Define the wavelet coefficients as floats
// Low coefficients
const std::vector<std::vector<float>> db_low = {
    {0.70710678f, 0.70710678f}, // db1
    {-0.12940952f, 0.22414387f, 0.83651630f, 0.48296291f}, // db2
    {0.03522629f, -0.08544127f, -0.13501102f, 0.45987750f, 0.80689151f, 0.33267055f}, // db3
    {-0.01059740f, 0.03288301f, 0.03084138f, -0.18703481f, -0.02798377f, 0.63088077f, 0.71484657f, 0.23037781f} // db4
};
// High coefficients
const std::vector<std::vector<float>> db_high = {
    {-0.70710678f, 0.70710678f}, // db1
    {-0.48296291f, 0.83651630f, -0.22414387f, -0.12940952f}, // db2
    {-0.33267055f, 0.80689151f, -0.45987750f, -0.13501102f, 0.08544127f, 0.03522629f}, // db3
    {-0.23037781f, 0.71484657f, -0.63088077f, -0.02798377f, 0.18703481f, 0.03084138f, -0.03288301f, -0.01059740f} // db4
};

void toGPU(std::vector<std::vector<std::vector<float>>> volume, int db_num, int depth, int rows, int cols, float*& d_low_coeff, float*& d_high_coeff, int&filter_size, float*& d_volume) {

    // Select the coefficients based on db_num
    std::vector<float> low_coeff = db_low[db_num - 1];
    std::vector<float> high_coeff = db_high[db_num - 1];

    // Calculate the filter size
    filter_size = low_coeff.size();

    // Allocate memory for the low and high pass coefficients on the GPU
    hipError_t err = hipMalloc(&d_low_coeff, low_coeff.size() * sizeof(float));
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate GPU memory for low coefficients: " + std::string(hipGetErrorString(err)));
    }

    err = hipMalloc(&d_high_coeff, high_coeff.size() * sizeof(float));
    if (err != hipSuccess) {
        hipFree(d_low_coeff); // Free previously allocated memory
        throw std::runtime_error("Failed to allocate GPU memory for high coefficients: " + std::string(hipGetErrorString(err)));
    }

    // Copy the coefficients to the GPU
    err = hipMemcpy(d_low_coeff, low_coeff.data(), low_coeff.size() * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(d_low_coeff);
        hipFree(d_high_coeff);
        throw std::runtime_error("Failed to copy low coefficients to GPU: " + std::string(hipGetErrorString(err)));
    }

    err = hipMemcpy(d_high_coeff, high_coeff.data(), high_coeff.size() * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(d_low_coeff);
        hipFree(d_high_coeff);
        throw std::runtime_error("Failed to copy high coefficients to GPU: " + std::string(hipGetErrorString(err)));
    }

    // Flatten the 3D volume into a 1D vector (row-major order)
    std::vector<float> flat_volume(depth * rows * cols);
    for (int d = 0; d < depth; ++d) {
        for (int r = 0; r < rows; ++r) {
            for (int c = 0; c < cols; ++c) {
                flat_volume[d * rows * cols + r * cols + c] = volume[d][r][c];
            }
        }
    }

    // Allocate memory on the GPU for the volume
    err = hipMalloc(&d_volume, flat_volume.size() * sizeof(float));
    if (err != hipSuccess) {
        hipFree(d_low_coeff);
        hipFree(d_high_coeff);
        throw std::runtime_error("Failed to allocate GPU memory for volume: " + std::string(hipGetErrorString(err)));
    }

    // Copy the flattened volume to the GPU
    err = hipMemcpy(d_volume, flat_volume.data(), flat_volume.size() * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(d_low_coeff);
        hipFree(d_high_coeff);
        hipFree(d_volume);
        throw std::runtime_error("Failed to copy volume data to GPU: " + std::string(hipGetErrorString(err)));
    }

    // Clear the CPU memory after copying to GPU
    flat_volume.clear();
    volume.clear();

    // Synchronize the device with error checking
    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to synchronize the device: " + std::string(hipGetErrorString(err)));
    }
}

std::vector<std::vector<std::vector<float>>> volCPU(float* d_volume, int depth, int rows, int cols) {
    // Allocate memory for the 3D volume on the CPU
    std::vector<std::vector<std::vector<float>>> volume(depth, std::vector<std::vector<float>>(rows, std::vector<float>(cols)));

    // Print the dimensions of the volume
    std::cerr << "Volume dimensions: " << depth << "x" << rows << "x" << cols << std::endl;

    // Copy the data from the GPU to the CPU
    std::vector<float> flat_volume(depth * rows * cols);
    hipError_t err = hipMemcpy(flat_volume.data(), d_volume, flat_volume.size() * sizeof(float), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        throw std::runtime_error("--Failed to copy data from GPU to CPU: " + std::string(hipGetErrorString(err)));
    }

    // Unflatten the 1D vector into a 3D volume
    for (int d = 0; d < depth; ++d) {
        for (int r = 0; r < rows; ++r) {
            for (int c = 0; c < cols; ++c) {
                volume[d][r][c] = flat_volume[(d * rows * cols) + (r * cols) + c];
            }
        }
    }

    // Free the allocated GPU memory
    err = hipFree(d_volume);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to free GPU memory: " + std::string(hipGetErrorString(err)));
    }

    return volume;
}

__global__ void copy_transformed_data(float* d_transformed, float* d_final, size_t depth, size_t rows, size_t cols, size_t orig_depth, size_t orig_rows, size_t orig_cols) {
    int d = blockIdx.z * blockDim.z + threadIdx.z;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (d < depth && r < rows && c < cols) {
        d_final[d * orig_rows * orig_cols + r * orig_cols + c] = d_transformed[d * rows * cols + r * cols + c];
    }
}


__global__ void extract_lll_subband(float* d_transformed, float* d_data, size_t depth, size_t rows, size_t cols) {
    int d = blockIdx.z * blockDim.z + threadIdx.z;
    int r = blockIdx.y * blockDim.y + threadIdx.y;
    int c = blockIdx.x * blockDim.x + threadIdx.x;

    if (d < depth / 2 && r < rows / 2 && c < cols / 2) {
        d_data[d * (rows / 2) * (cols / 2) + r * (cols / 2) + c] = d_transformed[d * rows * cols + r * cols + c];
    }
}


void multi_level(float* d_volume, float* d_low_coeff, float* d_high_coeff, int levels, int& depth, int& rows, int& cols, int filter_size) {
   
    // Allocate memory for the input data on the device
    float* d_data = d_volume;
    int data_size = depth * rows * cols * sizeof(float);

    // Allocate memory for the final transformed data on the device
    float* d_final;
    int init_depth = depth;
    int init_rows = rows;
    int init_cols = cols;
    hipError_t err = hipMalloc(&d_final, init_depth * init_rows * init_cols * sizeof(float));
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to allocate GPU memory for final data: " + std::string(hipGetErrorString(err)));
    }
    err = hipMemset(d_final, 0, init_depth * init_rows * init_cols * sizeof(float));
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to set final data to zero: " + std::string(hipGetErrorString(err)));
    }

    for (int level = 0; level < levels; ++level) {
        // Allocate memory for temporary data on the device
        float* d_temp;
        err = hipMalloc(&d_temp, data_size);
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for temporary data: " + std::string(hipGetErrorString(err)));
        }

        dim3 blockDim(16, 8, 8);
        dim3 gridDim0((rows + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y, (depth + blockDim.z - 1) / blockDim.z);
        dim3 gridDim1((cols + blockDim.x - 1) / blockDim.x, (rows + blockDim.y - 1) / blockDim.y, (depth + blockDim.z - 1) / blockDim.z);
        dim3 gridDim2((depth + blockDim.x - 1) / blockDim.x, (cols + blockDim.y - 1) / blockDim.y, (rows + blockDim.z - 1) / blockDim.z);

        // Perform convolution along the first dimension
        dim0_kernel<<<gridDim0, blockDim, filter_size * sizeof(float) * 2>>>(d_data, d_temp, d_low_coeff, d_high_coeff, filter_size, depth, rows, cols);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to synchronize device after dim0 kernel: " + std::string(hipGetErrorString(err)));
        }

        // Perform convolution along the second dimension
        dim1_kernel<<<gridDim1, blockDim, filter_size * sizeof(float) * 2>>>(d_temp, d_data, d_low_coeff, d_high_coeff, filter_size, depth, rows, cols);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to synchronize device after dim1 kernel: " + std::string(hipGetErrorString(err)));
        }

        // Perform convolution along the third dimension
        dim2_kernel<<<gridDim2, blockDim, filter_size * sizeof(float) * 2>>>(d_data, d_temp, d_low_coeff, d_high_coeff, filter_size, depth, rows, cols);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to synchronize device after dim2 kernel: " + std::string(hipGetErrorString(err)));
        }

        // Copy transformed data to the final array
        dim3 gridDim((cols + blockDim.x - 1) / blockDim.x, (rows + blockDim.y - 1) / blockDim.y, (depth + blockDim.z - 1) / blockDim.z);
        copy_transformed_data<<<gridDim, blockDim>>>(d_temp, d_final, depth, rows, cols, init_depth, init_rows, init_cols);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to synchronize device after copying transformed data: " + std::string(hipGetErrorString(err)));
        }

        // Update dimensions for the next level
        int new_depth = depth / 2;
        int new_rows = rows / 2;
        int new_cols = cols / 2;


        // Allocate memory for the new data on the device
        float* d_new_data;
        err = hipMalloc(&d_new_data, new_depth * new_rows * new_cols * sizeof(float));
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to allocate GPU memory for new data: " + std::string(hipGetErrorString(err)));
        }

        // Extract the LLL subband
        extract_lll_subband<<<gridDim, blockDim>>>(d_temp, d_new_data, depth, rows, cols);
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to synchronize device after extracting LLL subband: " + std::string(hipGetErrorString(err)));
        }

        // Free the old data and update pointers
        err = hipFree(d_data);
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to free GPU memory for old data: " + std::string(hipGetErrorString(err)));
        }
        d_data = d_new_data;
        data_size = new_depth * new_rows * new_cols * sizeof(float);

        depth = new_depth;
        rows = new_rows;
        cols = new_cols;

        // Free the temporary data
        err = hipFree(d_temp);
        if (err != hipSuccess) {
            throw std::runtime_error("Failed to free GPU memory for temporary data: " + std::string(hipGetErrorString(err)));
        }
    }
    // Swap the pointer to the final data
    err = hipFree(d_volume);
    if (err != hipSuccess) {
        throw std::runtime_error("Failed to free GPU memory for old data: " + std::string(hipGetErrorString(err)));
    }
    d_volume = d_final;
}


// Main program entry point
int main(int argc, char *argv[]) {
    (void)argc; // Suppress unused parameter warning
    // Print the program title
    std::cerr << "Assignment 2: CUDA Implementation of 3D DWT" << std::endl;

    // Check if the number of arguments is correct
    assert(argc == 5 && "Usage: ./assignment-2 <input.bin> <output.bin> <db_num> <levels>");

    // Start the global timer
    auto start = std::chrono::high_resolution_clock::now();

    // Load the arguments into variables
    std::string bin_in = argv[1];
    std::string bin_out = argv[2];
    int db_num = std::stoi(argv[3]);
    int levels = std::stoi(argv[4]);

    // Load the 3D volume from the binary file
    std::vector<std::vector<std::vector<float>>> vol_in = loadvolume(bin_in);

    // Get the dimensions of the 3D volume
    int depth = vol_in.size();
    int rows = vol_in[0].size();
    int cols = vol_in[0][0].size();

    // Print the dimensions of the 3D volume
    std::cerr << "Volume dimensions: " << depth << "x" << rows << "x" << cols << std::endl;

    // Create pointers for the volume and coefficients on the GPU
    float* d_low_coeff = nullptr;
    float* d_high_coeff = nullptr;
    float* d_volume = nullptr;
    int filter_size;

    // Copy the volume and coefficients to the GPU
    toGPU(vol_in, db_num, depth, rows, cols, d_low_coeff, d_high_coeff, filter_size, d_volume);
    
    // Perform the multi-level DWT on the 3D volume
    multi_level(d_volume, d_low_coeff, d_high_coeff, levels, depth, rows, cols, filter_size);

    // Copy the data back to the CPU
    std::vector<std::vector<std::vector<float>>> vol_out = volCPU(d_volume, depth, rows, cols);

    // print the dimensions of volume after DWT
    std::cerr << "Volume dimensions after DWT: " << vol_out.size() << "x" << vol_out[0].size() << "x" << vol_out[0][0].size() << std::endl;

    // Save the modified 3D volume to the output binary file
    savevolume(vol_out, bin_out);

    // Stop the global timer
    auto end = std::chrono::high_resolution_clock::now();
    // Calculate the duration
    std::chrono::duration<double> t = end - start;

    // Log the time taken for the program
    std::cerr << "Total time taken: " << t.count() << " seconds" << std::endl;

    return EXIT_SUCCESS;
}